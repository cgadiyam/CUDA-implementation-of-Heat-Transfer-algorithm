#include "hip/hip_runtime.h"
/************************************************************************/
// The purpose of this file is to provide a GPU implementation of the 
// heat transfer simulation using MATLAB.
//
// Author: Jason Lowden
// Date: October 20, 2013
//
// File: KMeans.h
/************************************************************************/
#include <hip/hip_runtime_api.h>
#include <>
#include <>
#include <hip/hip_texture_types.h>
#include <iostream>
#include "HeatTransfer.h"

texture<float, hipTextureType2D, hipReadModeElementType> heatTexture;

__global__ void UpdateHeatMapKernel(float *OutputData, int size, float heatSpeed)
{
	int x=threadIdx.x + blockIdx.x*blockDim.x;
	int y=threadIdx.y + blockIdx.y*blockDim.y;
	int tID = x+(y*size);
	float t_center, t_left, t_right, t_bottom, t_top;
	if(x>0 && x<(size-1) && y>0 && y<(size-1))
	{
		t_top= tex2D(heatTexture,x,y-1);
		t_left= tex2D(heatTexture,x-1,y);
		t_center= tex2D(heatTexture,x,y);
		t_right= tex2D(heatTexture,x+1,y);
		t_bottom= tex2D(heatTexture,x,y+1); 
		OutputData[tID] = t_center + ((t_top + t_left + t_right + t_bottom - (4 * t_center)) * heatSpeed);
	}

}

bool UpdateHeatMap(float* dataIn, float* dataOut, int size, float heatSpeed, int numIterations)
{
	hipError_t status;
	float *OutputData;
	hipArray_t InputData;
	int bytes = size * size * sizeof(float);
	hipMalloc((void**) &OutputData, bytes);
	hipMemcpy(OutputData, dataIn, (size*size*sizeof(float)), hipMemcpyHostToDevice);
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipMallocArray (&InputData, &desc, size, size, 0);
	hipMemcpyToArray (InputData, 0, 0, dataIn, (size*size*sizeof(float)),  hipMemcpyHostToDevice);
	hipBindTextureToArray (&heatTexture, InputData, &desc);
	for(int i=0;i<numIterations;i++)
	{
		dim3 dimBlock(16, 16); 
		dim3 dimGrid((int)ceil((float)size/16), (int)ceil((float)size/16));
		UpdateHeatMapKernel<<<dimGrid, dimBlock>>>(OutputData, size , heatSpeed);
		hipDeviceSynchronize();
		// Check for errors
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
			std::cout << "Kernel failed: " << hipGetErrorString(status) << std::endl;
			hipUnbindTexture (&heatTexture);
			hipFree(OutputData);
			return false;
		}
		hipUnbindTexture (&heatTexture);
		hipMemcpyToArray (InputData, 0, 0, OutputData, (size*size*sizeof(float)),  hipMemcpyDeviceToDevice);
		hipBindTextureToArray (&heatTexture, InputData, &desc);
	}

	hipMemcpy(dataOut, OutputData, (size*size*sizeof(float)), hipMemcpyDeviceToHost);
	hipFree(OutputData);
	hipUnbindTexture (&heatTexture);
	hipFreeArray(InputData);
	return true;
}